

#include <stdlib.h>
#include <stdio.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


 
 /**
 Demag matrices to be stored in GPU memory
 */
 float* d_Kxx = NULL;
 float* d_Kxy = NULL;
 float* d_Kxz = NULL;
 float* d_Kyy = NULL;
 float* d_Kyz = NULL;
 float* d_Kzz = NULL;
 
 
 //Magnetization vectors on GPU
 float* d_Mx = NULL;
 float* d_My = NULL;
 float* d_Mz = NULL;
 
 //Magnetic field vectors on GPU
 float* d_Hx = NULL;
 float* d_Hy = NULL;
 float* d_Hz = NULL;
 
 //size of the matrices (n_K x n_K)
 int n_K;
 
 /**
 Sparse demag matrices to be stored in the GPU memory
 */
 
 struct CUSparse
 {

	//data values
	float* values = NULL;
	//column indices (size nnz)
	int* cols;
	//row indices (size n + 1  with the last element equal to nnz)
	int* rows;
	//no of matrix elements (matrices are n x n )
	int n;
	//no. of non zero element
	int nnz;
 };
	
 
 CUSparse spKxx;
 CUSparse spKxy;
 CUSparse spKxz;
 CUSparse spKyy;
 CUSparse spKyz;
 CUSparse spKzz;
 
 //general handle for cuBlas (initalized once)
 hipblasHandle_t handle = NULL;
 
 
 //handle to the sparse matrix description
hipsparseMatDescr_t sparse_descr = NULL;

//handle to the sparse matrix blas
hipsparseHandle_t sparse_handle = NULL; 
 
 void loadSparseToDevice( float* values, int* colInds, int* rowInds, float* d_values, int* d_colInds, int* d_rowInds, hipsparseSpMatDescr_t mat, int nnz_ );
 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, 2019
 Initializes the sparse matrices for later use in the matrix-vector multiplications
 n is the no. of rows and cols int he matrices
 nnz is the no. of non-zero elements
 mat_no 1...6 identifies which matrix to load (Kxx = 1, Kxy = 2, Kxz = 3, Kyy = 4, Kyz = 5, Kzz = 6 )
 values float array with the non-zero values of the matrix (length nnz)
 colInds int array with the column indices of the CSR stored matrix (length nnz)
 rowInds the indices of the first non-zero element in each row (size nrows+1 and the last element has to be equal to nnz + rowInds[0] )
 */
 void cu_initDemagMatrices_sparse( const int* n, const int* nnz, const int* mat_no, float* values, int* colInds, int* rowInds )
 {
	
	
	 //should only be done once
	 if ( sparse_handle == NULL )
	 {
		 //handle to the sparse matrix multiplier in CUDA
		hipsparseStatus_t status = hipsparseCreate(&sparse_handle);
		
		//no. of rows and columns
		spKxx.n = *n;
		spKxy.n = *n;
		spKxz.n = *n;
		spKyy.n = *n;
		spKyz.n = *n;
		spKzz.n = *n;
		
		//allocate memory for the magnetization vectors
		hipMalloc((void**) &d_Mx, *n * sizeof(float));
		hipMalloc((void**) &d_My, *n * sizeof(float));
		hipMalloc((void**) &d_Mz, *n * sizeof(float));
		
		//allocate memory for the magnetic field vectors
		hipMalloc((void**) &d_Hx, *n * sizeof(float));
		hipMalloc((void**) &d_Hy, *n * sizeof(float));
		hipMalloc((void**) &d_Hz, *n * sizeof(float));
			 
		status = hipsparseCreateMatDescr(&sparse_descr);
		hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ONE);
		
		/*Kaspar: This code will not work on Windows before the next major release thanks to nvidia
		//The handle on the device for the magnetization vectors
		hipsparseCreateDnVec( &d_Mx_descr, n_sparse, d_Mx, HIP_R_32F);
		hipsparseCreateDnVec( &d_My_descr, n_sparse, d_My, HIP_R_32F);
		hipsparseCreateDnVec( &d_Mz_descr, n_sparse, d_Mz, HIP_R_32F);
		
		//handles for the field on the device
		hipsparseCreateDnVec( &d_Hx_descr, n_sparse, d_Hx, HIP_R_32F);
		hipsparseCreateDnVec( &d_Hy_descr, n_sparse, d_Hy, HIP_R_32F);
		hipsparseCreateDnVec( &d_Hz_descr, n_sparse, d_Hz, HIP_R_32F);
			*/	
		
		
	 }
	
	switch( *mat_no )
	{
		case 1:
			
			loadSparseToDevice( values, colInds, rowInds, spKxx, *nnz );
								
			break;
		case 2:
			loadSparseToDevice( values, colInds, rowInds, spKxy, *nnz );
			
			break;
		case 3:
			loadSparseToDevice( values, colInds, rowInds, spKxz, *nnz );
			
			break;
		case 4:
			loadSparseToDevice( values, colInds, rowInds, spKyy, *nnz );
			
			break;
		case 5:
			loadSparseToDevice( values, colInds, rowInds, spKyz, *nnz );
			
			break;
		case 6:
			loadSparseToDevice( values, colInds, rowInds, spKzz, *nnz );
			
			break;
			
	}
	
	
	
 }
 
 
 void loadSparseToDevice( float* values, int* colInds, int* rowInds, float* d_values, CUSparse* mat, int nnz_ )
 {
	 
	 //allocate the row inds (+1 in size as the last element contains nnz + rowInds(0)
	hipMalloc((void**) mat->rows, (mat->n + 1) * sizeof(int));
	//allocate the column inds
	hipMalloc((void**) mat->cols, mat->nnz * sizeof(int));
	//allocate the values array
	hipMalloc((void**) mat->values, mat->nnz * sizeof(float));
	
	//copy to device
	hipMemcpy( mat->rows, rowInds, (mat->n+1) * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( mat->cols, colInds, mat->nnz * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy( mat->values, values, mat->nnz * sizeof(float),hipMemcpyHostToDevice);
	
	
	
	//init the sparse matrix handles
	/*Kaspar: This code will not work before the next major release thanks to nvidia
	hipsparseCreateCsr(&mat, n_sparse, n_sparse, nnz_, d_rowInds, d_colInds, d_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F);
									  */
 }
 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, 2019
 Does the matrix-vector product of a sparse matrix and a dense vector
 and returns the demag field on the form:
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kyz * Mx + Kyy * My + Kzz * Mz
 Hz = Kzx * Mx + Kzy * My + Kzz * Mz
 
 */
 void cu_MVMult_GetH_sparse(const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, const float* pref)
 {	 
	
	hipsparseStatus_t stat;
	
	
	void* dBuffer = NULL;
	int n = spKxx.n;
	//Copy Mx, My and Mz to device memory	
	hipMemcpy( d_Mx, Mx, n * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_My, My, n * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_Mz, Mz, n * sizeof(float), hipMemcpyHostToDevice );
	
	float alpha = *pref;
	float beta = 0.0;
	//by setting beta = 0 in the first call we ensure that the previous values of d_Hx are irrelevant
	//as this operation does the following:
	//d_Hx = alpha * d_Kxx * d_Mx + beta * d_Hx
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKxx.nnz,
                           &alpha, sparse_descr, spKxx.values, spKxx.rows, spKxx.cols,
                           d_Mx, &beta, d_Hx);
	
	//kaspar: All the calls below will not work before the next major release by nvidia
	
	//Hx = Kxx * Mx + Kxy * My + Kxz * Mz	
	//Kxx * Mx
	/*
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxx_descr, d_Mx_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			
	beta = 1.0;
	//Kxy * My
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxy_descr, d_My_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxz_descr, d_Mz_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

			   
	//Hy = Kxy * Mx + Kyy * My + Kyz * Mz	
	beta = 0.;
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxy_descr, d_Mx_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			
	beta = 1.0;	
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kyy_descr, d_My_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kyz_descr, d_Mz_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	//Hz = Kzx * Mx + Kzy * My + Kzz * Mz		
	beta = 0.;
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxz_descr, d_Mx_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			
	beta = 1.0;
	//Kxy * My
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kyz_descr, d_My_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kzz_descr, d_Mz_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			   
			   */
	//copy the solution back
	hipMemcpy( Hx, d_Hx, n * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( Hy, d_Hy, n * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( Hz, d_Hz, n * sizeof(float), hipMemcpyDeviceToHost );
 }
 
 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, DTU 2019
 Method for transfering the demagnetization matrices from Fortran
 @param Kxx, Kxy, Kxz, Kyy, Kyz and Kzz are 2D arrays of size n x n implemented as an array of n pointers each pointing to an array of n values
 @param n is the size of the square matrix
 */
 void cu_initDemagMatrices( const float* Kxx, const float* Kxy, const float* Kxz, const float* Kyy, const float* Kyz, const float* Kzz, int* n)
 {	 
	/* FILE *fp;

	fp = fopen("test.txt", "w+");
	fprintf(fp,"%f %d\n", Kxx[0], *n );
	//fprintf(fp, "This is testing for fprintf...\n");
	//fputs("This is testing for fputs...\n", fp);
	fclose(fp);
	 */
	 
	 hipblasStatus_t stat;
	 n_K = *n;
	 
	 //Allocate the device (GPU) arrays
	 size_t bytes = n_K * n_K * sizeof(float);
	 
	 hipMalloc( &d_Kxx, bytes );
	 hipMalloc( &d_Kxy, bytes );
	 hipMalloc( &d_Kxz, bytes );
	 hipMalloc( &d_Kyy, bytes );
	 hipMalloc( &d_Kyz, bytes );
	 hipMalloc( &d_Kzz, bytes );
	 
	 
	 
	 //copy the demag tensors to the device	 
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxx, n_K, d_Kxx, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxy, n_K, d_Kxy, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxz, n_K, d_Kxz, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kyy, n_K, d_Kyy, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kyz, n_K, d_Kyz, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kzz, n_K, d_Kzz, n_K );
	 
	//allocate the internal M and H vectors
	bytes = n_K * sizeof(float);
	hipMalloc( &d_Mx, bytes );
	hipMalloc( &d_My, bytes );
	hipMalloc( &d_Mz, bytes );

	hipMalloc( &d_Hx, bytes );
	hipMalloc( &d_Hy, bytes );
	hipMalloc( &d_Hz, bytes );
	
	//initialize the cuBlas handle
	hipblasCreate(&handle);
	 
 }
 
 /**
 Get the resulting demag field through
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kxy * Mx + Kyy * My + Kyz * Mz
 Hz = Kxz * Mx + Kyz * My + Kzz * Mz
 
 Noting that K is symmetric, i.e. Kxy = Kyx etc.
 
 @param Mx, My and Mz are n,1 arrays assuming that n == n_K (no checking, so use it correctly!)
 @param Hx, Hy and Hz are the output field vectors (also size n,1)
 @param pref is the factor that goes in front of the multiplication, i.e.
 Hx = pref * ( Kxx * Mx + Kxy * My + Kxz * Mz ) etc.
 */
 void cu_MVMult_GetH( const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, int* n, float* pref )
 {
	 hipblasStatus_t stat;
	 //copy the M vectors to the GPU card
	 stat = hipblasSetVector (n_K, sizeof (float), Mx, 1, d_Mx, 1);
	 stat = hipblasSetVector (n_K, sizeof (float), My, 1, d_My, 1);
	 stat = hipblasSetVector (n_K, sizeof (float), Mz, 1, d_Mz, 1);
	 
	 float beta = 0.;
	 
	 //Kxx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxx, n_K, d_Mx, 1, &beta, d_Hx, 1);
	 //Kxy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_My, 1, &beta, d_Hx, 1);
	 //Kxz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mz, 1, &beta, d_Hx, 1);
	 
	 beta = 0.;
	 //Kyx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_Mx, 1, &beta, d_Hy, 1);
	 //Kyy * My
	 beta = 1.; //change beta = 1 so that Hy is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyy, n_K, d_My, 1, &beta, d_Hy, 1);
	 //Kyz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_Mz, 1, &beta, d_Hy, 1);
	 
	 beta = 0.;
	 //Kzx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mx, 1, &beta, d_Hz, 1);
	 //Kzy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_My, 1, &beta, d_Hz, 1);
	 //Kzz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kzz, n_K, d_Mz, 1, &beta, d_Hz, 1);
	 
	 //copy the resulting field vector back	 
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hx, 1, Hx, 1);
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hy, 1, Hy, 1);
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hz, 1, Hz, 1);
	 
 }
 
 void cu_test( const float* K, int* n)
 {
	 // FILE *fp;

	//fp = fopen("test.txt", "w+");
	for ( int i=0; i < *n; i++ )
		printf("%f %d\n", K[i], 12 );
	
	//fclose(fp);
 }
 
 void cu_destroy()
 {
	 if ( d_Kxx != NULL )
	 {
		hipFree( d_Kxx );
		 hipFree( d_Kxy );
		 hipFree( d_Kxz );
		 hipFree( d_Kyy );
		 hipFree( d_Kyz );
		 hipFree( d_Kzz );
	 }
	 
	 if ( d_Hx != NULL )
	 {	 
		 hipFree( d_Hx );
		 hipFree( d_Hy );
		 hipFree( d_Hz );
	 }
	 
	 if ( d_Mx != NULL )
	 {
		 hipFree( d_Mx );
		 hipFree( d_My );
		 hipFree( d_Mz );
	 }
	 
	 if ( handle != NULL )
		hipblasDestroy(handle);
	
	if (sparse_handle != NULL )
	{
		hipsparseDestroy(sparse_handle);
	    hipFree( spKxx.values );
		hipFree( spKxy.values );
		hipFree( spKxz.values );
		hipFree( spKyy.values );
		hipFree( spKyz.values );
		hipFree( spKzz.values );
		
		hipFree( spKxx.cols );
		hipFree( spKxy.cols );
		hipFree( spKxz.cols );
		hipFree( spKyy.cols );
		hipFree( spKyz.cols );
		hipFree( spKzz.cols );
		
		hipFree( spKxx.rows );
		hipFree( spKxy.rows );
		hipFree( spKxz.rows );
		hipFree( spKyy.rows );
		hipFree( spKyz.rows );
		hipFree( spKzz.rows );
	}

 }