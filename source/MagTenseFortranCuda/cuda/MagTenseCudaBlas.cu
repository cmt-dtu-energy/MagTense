

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include "hipblas.h"



 
 /**
 Demag matrices to be stored in GPU memory
 */
 float* d_Kxx;
 float* d_Kxy;
 float* d_Kxz;
 float* d_Kyy;
 float* d_Kyz;
 float* d_Kzz;
 
 //Magnetization vectors on GPU
 float* d_Mx;
 float* d_My;
 float* d_Mz;
 
 //Magnetic field vectors on GPU
 float* d_Hx;
 float* d_Hy;
 float* d_Hz;
 
 //size of the matrices (n_K x n_K)
 int n_K;
 
 //general handle for cuBlas (initalized once)
 hipblasHandle_t handle;
 
 /**
 Kaspar K. Nielsen, kaki@dtu.dk, DTU 2019
 Method for transfering the demagnetization matrices from Fortran
 @param Kxx, Kxy, Kxz, Kyy, Kyz and Kzz are 2D arrays of size n x n implemented as an array of n pointers each pointing to an array of n values
 @param n is the size of the square matrix
 */
 void cu_initDemagMatrices( const float* Kxx, const float* Kxy, const float* Kxz, const float* Kyy, const float* Kyz, const float* Kzz, int* n)
 {
	 FILE *fp;

	fp = fopen("test.txt", "w+");
	fprintf(fp,"%f %d\n", Kxx[0], *n );
	//fprintf(fp, "This is testing for fprintf...\n");
	//fputs("This is testing for fputs...\n", fp);
	fclose(fp);
	 
	 
	 hipblasStatus_t stat;
	 n_K = *n;
	 
	 //Allocate the device (GPU) arrays
	 size_t bytes = n_K * n_K * sizeof(float);
	 
	 hipMalloc( &d_Kxx, bytes );
	 hipMalloc( &d_Kxy, bytes );
	 hipMalloc( &d_Kxz, bytes );
	 hipMalloc( &d_Kyy, bytes );
	 hipMalloc( &d_Kyz, bytes );
	 hipMalloc( &d_Kzz, bytes );
	 
	 
	 
	 //copy the demag tensors to the device	 
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxx, n_K, d_Kxx, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxy, n_K, d_Kxy, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxz, n_K, d_Kxz, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kyy, n_K, d_Kyy, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kyz, n_K, d_Kyz, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kzz, n_K, d_Kzz, n_K );
	 
	//allocate the internal M and H vectors
	bytes = n_K * sizeof(float);
	hipMalloc( &d_Mx, bytes );
	hipMalloc( &d_My, bytes );
	hipMalloc( &d_Mz, bytes );

	hipMalloc( &d_Hx, bytes );
	hipMalloc( &d_Hy, bytes );
	hipMalloc( &d_Hz, bytes );
	
	//initialize the cuBlas handle
	hipblasCreate(&handle);
	 
 }
 
 /**
 Get the resulting demag field through
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kxy * Mx + Kyy * My + Kyz * Mz
 Hz = Kxz * Mx + Kyz * My + Kzz * Mz
 
 Noting that K is symmetric, i.e. Kxy = Kyx etc.
 
 @param Mx, My and Mz are n,1 arrays assuming that n == n_K (no checking, so use it correctly!)
 @param Hx, Hy and Hz are the output field vectors (also size n,1)
 @param pref is the factor that goes in front of the multiplication, i.e.
 Hx = pref * ( Kxx * Mx + Kxy * My + Kxz * Mz ) etc.
 */
 void cu_MVMult_GetH( const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, int* n, float* pref )
 {
	 hipblasStatus_t stat;
	 //copy the M vectors to the GPU card
	 stat = hipblasSetVector (n_K, sizeof (float), Mx, 1, d_Mx, 1);
	 stat = hipblasSetVector (n_K, sizeof (float), My, 1, d_My, 1);
	 stat = hipblasSetVector (n_K, sizeof (float), Mz, 1, d_Mz, 1);
	 
	 float beta = 0.;
	 
	 //Kxx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxx, n_K, d_Mx, 1, &beta, d_Hx, 1);
	 //Kxy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_My, 1, &beta, d_Hx, 1);
	 //Kxz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mz, 1, &beta, d_Hx, 1);
	 
	 beta = 0.;
	 //Kyx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_Mx, 1, &beta, d_Hy, 1);
	 //Kyy * My
	 beta = 1.; //change beta = 1 so that Hy is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyy, n_K, d_My, 1, &beta, d_Hy, 1);
	 //Kyz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_Mz, 1, &beta, d_Hy, 1);
	 
	 beta = 0.;
	 //Kzx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mx, 1, &beta, d_Hz, 1);
	 //Kzy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_My, 1, &beta, d_Hz, 1);
	 //Kzz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kzz, n_K, d_Mz, 1, &beta, d_Hz, 1);
	 
	 //copy the resulting field vector back	 
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hx, 1, Hx, 1);
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hy, 1, Hy, 1);
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hz, 1, Hz, 1);
	 
 }
 
 void cu_test( const float* K, int* n)
 {
	 // FILE *fp;

	//fp = fopen("test.txt", "w+");
	for ( int i=0; i < *n; i++ )
		printf("%f %d\n", K[i], 12 );
	
	//fclose(fp);
 }
 
 void cu_destroy()
 {
	 hipFree( d_Kxx );
	 hipFree( d_Kxy );
	 hipFree( d_Kxz );
	 hipFree( d_Kyy );
	 hipFree( d_Kyz );
	 hipFree( d_Kzz );
	 
	 hipFree( d_Hx );
	 hipFree( d_Hy );
	 hipFree( d_Hz );
	 
	 hipFree( d_Mx );
	 hipFree( d_My );
	 hipFree( d_Mz );
	 
	 hipblasDestroy(handle);
 }