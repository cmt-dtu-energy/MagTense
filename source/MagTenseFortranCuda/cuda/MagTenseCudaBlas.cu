

#include <stdlib.h>
#include <stdio.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


 
 /**
 Demag matrices to be stored in GPU memory
 */
 float* d_Kxx = NULL;
 float* d_Kxy = NULL;
 float* d_Kxz = NULL;
 float* d_Kyy = NULL;
 float* d_Kyz = NULL;
 float* d_Kzz = NULL;
 
 
 //Magnetization vectors on GPU
 float* d_Mx = NULL;
 float* d_My = NULL;
 float* d_Mz = NULL;
 
 //Magnetic field vectors on GPU
 float* d_Hx = NULL;
 float* d_Hy = NULL;
 float* d_Hz = NULL;
 
 //size of the matrices (n_K x n_K)
 int n_K;
 
 /**
 Sparse demag matrices to be stored in the GPU memory
 */
 
 struct CUSparse
 {

	//data values
	float* values = NULL;
	//column indices (size nnz)
	int* cols;
	//row indices (size n + 1  with the last element equal to nnz)
	int* rows;
	//no of matrix elements (matrices are n x n )
	int n;
	//no. of non zero element
	int nnz;
 };
	
 
 CUSparse spKxx;
 CUSparse spKxy;
 CUSparse spKxz;
 CUSparse spKyy;
 CUSparse spKyz;
 CUSparse spKzz;
 
 //general handle for cuBlas (initalized once)
 hipblasHandle_t handle = NULL;
 
 
 //handle to the sparse matrix description
hipsparseMatDescr_t sparse_descr = NULL;

//handle to the sparse matrix blas
hipsparseHandle_t sparse_handle = NULL; 
 
 void loadSparseToDevice( float* values, int* colInds, int* rowInds, CUSparse* mat, int nnz_ );
 void freeSparseMatrix( CUSparse* mat );
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, 2019
 Initializes the sparse matrices for later use in the matrix-vector multiplications
 n is the no. of rows and cols in the matrices
 nnz is the no. of non-zero elements
 mat_no 1...6 identifies which matrix to load (Kxx = 1, Kxy = 2, Kxz = 3, Kyy = 4, Kyz = 5, Kzz = 6 )
 values float array with the non-zero values of the matrix (length nnz)
 colInds int array with the column indices of the CSR stored matrix (length nnz)
 rowInds the indices of the first non-zero element in each row (size nrows+1 and the last element has to be equal to nnz + rowInds[0] )
 */
 void cu_initDemagMatrices_sparse( const int* n, const int* nnz, const int* mat_no, float* values, int* colInds, int* rowInds )
 {
	
	
	 //should only be done once
	 if ( sparse_handle == NULL )
	 {
		 //handle to the sparse matrix multiplier in CUDA
		hipsparseStatus_t status = hipsparseCreate(&sparse_handle);
		
		//no. of rows and columns
		spKxx.n = *n;
		spKxy.n = *n;
		spKxz.n = *n;
		spKyy.n = *n;
		spKyz.n = *n;
		spKzz.n = *n;
		hipError_t err;
		//allocate memory for the magnetization vectors
		err = hipMalloc((void**) &d_Mx, *n * sizeof(float));
		err = hipMalloc((void**) &d_My, *n * sizeof(float));
		err = hipMalloc((void**) &d_Mz, *n * sizeof(float));
		
		//allocate memory for the magnetic field vectors
		err = hipMalloc((void**) &d_Hx, *n * sizeof(float));
		err = hipMalloc((void**) &d_Hy, *n * sizeof(float));
		err = hipMalloc((void**) &d_Hz, *n * sizeof(float));
			 
		status = hipsparseCreateMatDescr(&sparse_descr);
		hipsparseSetMatType(sparse_descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(sparse_descr,HIPSPARSE_INDEX_BASE_ONE);
		
		/*Kaspar: This code will not work on Windows before the next major release thanks to nvidia
		//The handle on the device for the magnetization vectors
		hipsparseCreateDnVec( &d_Mx_descr, n_sparse, d_Mx, HIP_R_32F);
		hipsparseCreateDnVec( &d_My_descr, n_sparse, d_My, HIP_R_32F);
		hipsparseCreateDnVec( &d_Mz_descr, n_sparse, d_Mz, HIP_R_32F);
		
		//handles for the field on the device
		hipsparseCreateDnVec( &d_Hx_descr, n_sparse, d_Hx, HIP_R_32F);
		hipsparseCreateDnVec( &d_Hy_descr, n_sparse, d_Hy, HIP_R_32F);
		hipsparseCreateDnVec( &d_Hz_descr, n_sparse, d_Hz, HIP_R_32F);
			*/	
		
		
	 }
	
	switch( *mat_no )
	{
		case 1:
			
			loadSparseToDevice( values, colInds, rowInds, &spKxx, *nnz );
								
			break;
		case 2:
			loadSparseToDevice( values, colInds, rowInds, &spKxy, *nnz );
			
			break;
		case 3:
			loadSparseToDevice( values, colInds, rowInds, &spKxz, *nnz );
			
			break;
		case 4:
			loadSparseToDevice( values, colInds, rowInds, &spKyy, *nnz );
			
			break;
		case 5:
			loadSparseToDevice( values, colInds, rowInds, &spKyz, *nnz );
			
			break;
		case 6:
			loadSparseToDevice( values, colInds, rowInds, &spKzz, *nnz );
			
			break;
			
	}
	
	
	
 }
 
 
 void loadSparseToDevice( float* values, int* colInds, int* rowInds, CUSparse* mat, int nnz_ )
 {
	hipError_t err;
	//set the no. of non-zero entries
 	mat->nnz = nnz_;
	 //allocate the row inds (+1 in size as the last element contains nnz + rowInds(0)
	err = hipMalloc((void**) &(mat->rows), (mat->n + 1) * sizeof(int));
	//allocate the column inds
	err = hipMalloc((void**) &(mat->cols), mat->nnz * sizeof(int));
	//allocate the values array
	err = hipMalloc((void**) &(mat->values), mat->nnz * sizeof(float));
	
	//copy to device
	err = hipMemcpy( mat->rows, rowInds, (mat->n+1) * sizeof(int),hipMemcpyHostToDevice);
	err = hipMemcpy( mat->cols, colInds, mat->nnz * sizeof(int),hipMemcpyHostToDevice);
	err = hipMemcpy( mat->values, values, mat->nnz * sizeof(float),hipMemcpyHostToDevice);
	
	
	
	//init the sparse matrix handles
	/*Kaspar: This code will not work before the next major release thanks to nvidia
	hipsparseCreateCsr(&mat, n_sparse, n_sparse, nnz_, d_rowInds, d_colInds, d_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F);
									  */
 }
 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, 2019
 Does the matrix-vector product of a sparse matrix and a dense vector
 and returns the demag field on the form:
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kyz * Mx + Kyy * My + Kzz * Mz
 Hz = Kzx * Mx + Kzy * My + Kzz * Mz
 
 */
 void cu_MVMult_GetH_sparse(const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, const float* pref)
 {	 
	
	hipsparseStatus_t status;
	
	
	
	int n = spKxx.n;
	//Copy Mx, My and Mz to device memory	
	hipMemcpy( d_Mx, Mx, n * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_My, My, n * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( d_Mz, Mz, n * sizeof(float), hipMemcpyHostToDevice );
	
	float alpha = *pref;
	float beta = 0.0;
	//by setting beta = 0 in the first call we ensure that the previous values of d_Hx are irrelevant
	//as this operation does the following:
	//d_Hx = alpha * d_Kxx * d_Mx + beta * d_Hx
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKxx.nnz,
                           &alpha, sparse_descr, spKxx.values, spKxx.rows, spKxx.cols,
                           d_Mx, &beta, d_Hx);
	
	//set beta = 1 so as to keep the previous set values of d_Hx
	beta = 1.0;
	//Hx = Kxy * My
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKxy.nnz,
                           &alpha, sparse_descr, spKxy.values, spKxy.rows, spKxy.cols,
                           d_My, &beta, d_Hx);
	//Hx = Kxz * Mz
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKxz.nnz,
                           &alpha, sparse_descr, spKxz.values, spKxz.rows, spKxz.cols,
                           d_Mz, &beta, d_Hx);						   
						   
						   
	//reset beta = 0 such that any previous values of d_Hy are erased
	beta = 0.;
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKxy.nnz,
                           &alpha, sparse_descr, spKxy.values, spKxy.rows, spKxy.cols,
                           d_Mx, &beta, d_Hy);
	
	//set beta = 1 so as to keep the previous set values of d_Hy
	beta = 1.0;
	//Hy = Kyy * My
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKyy.nnz,
                           &alpha, sparse_descr, spKyy.values, spKyy.rows, spKyy.cols,
                           d_My, &beta, d_Hy);
	//Hy = Kyz * Mz
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKyz.nnz,
                           &alpha, sparse_descr, spKyz.values, spKyz.rows, spKyz.cols,
                           d_Mz, &beta, d_Hy);						   
						   
						   
	//reset beta = 0 such that any previous values of d_Hz are erased
	beta = 0.;
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKxz.nnz,
                           &alpha, sparse_descr, spKxz.values, spKxz.rows, spKxz.cols,
                           d_Mx, &beta, d_Hz);
	
	//set beta = 1 so as to keep the previous set values of d_Hy
	beta = 1.0;
	//Hz = Kzy * My
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKyz.nnz,
                           &alpha, sparse_descr, spKyz.values, spKyz.rows, spKyz.cols,
                           d_My, &beta, d_Hz);
	//Hz = Kzz * Mz
	status = hipsparseScsrmv( sparse_handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, n, n, spKzz.nnz,
                           &alpha, sparse_descr, spKzz.values, spKzz.rows, spKzz.cols,
                           d_Mz, &beta, d_Hz);									   
						   
	//kaspar: All the calls below will not work before the next major release by nvidia
	
	//Hx = Kxx * Mx + Kxy * My + Kxz * Mz	
	//Kxx * Mx
	/*
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxx_descr, d_Mx_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			
	beta = 1.0;
	//Kxy * My
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxy_descr, d_My_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxz_descr, d_Mz_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);

			   
	//Hy = Kxy * Mx + Kyy * My + Kyz * Mz	
	beta = 0.;
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxy_descr, d_Mx_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			
	beta = 1.0;	
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kyy_descr, d_My_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kyz_descr, d_Mz_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	//Hz = Kzx * Mx + Kzy * My + Kzz * Mz		
	beta = 0.;
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kxz_descr, d_Mx_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			
	beta = 1.0;
	//Kxy * My
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kyz_descr, d_My_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
								 
	hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, d_Kzz_descr, d_Mz_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_MV_ALG_DEFAULT, dBuffer);
			   
			   */
	//copy the solution back
	hipMemcpy( Hx, d_Hx, n * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( Hy, d_Hy, n * sizeof(float), hipMemcpyDeviceToHost );
	hipMemcpy( Hz, d_Hz, n * sizeof(float), hipMemcpyDeviceToHost );
 }
 
 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, DTU 2019
 Method for transfering the demagnetization matrices from Fortran
 @param Kxx, Kxy, Kxz, Kyy, Kyz and Kzz are 2D arrays of size n x n implemented as an array of n pointers each pointing to an array of n values
 @param n is the size of the square matrix
 */
 void cu_initDemagMatrices( const float* Kxx, const float* Kxy, const float* Kxz, const float* Kyy, const float* Kyz, const float* Kzz, int* n)
 {	 
	/* FILE *fp;

	fp = fopen("test.txt", "w+");
	fprintf(fp,"%f %d\n", Kxx[0], *n );
	//fprintf(fp, "This is testing for fprintf...\n");
	//fputs("This is testing for fputs...\n", fp);
	fclose(fp);
	 */
	 
	 hipblasStatus_t stat;
	 n_K = *n;
	 
	 //Allocate the device (GPU) arrays
	 size_t bytes = n_K * n_K * sizeof(float);
	 
	 hipMalloc( &d_Kxx, bytes );
	 hipMalloc( &d_Kxy, bytes );
	 hipMalloc( &d_Kxz, bytes );
	 hipMalloc( &d_Kyy, bytes );
	 hipMalloc( &d_Kyz, bytes );
	 hipMalloc( &d_Kzz, bytes );
	 
	 
	 
	 //copy the demag tensors to the device	 
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxx, n_K, d_Kxx, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxy, n_K, d_Kxy, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kxz, n_K, d_Kxz, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kyy, n_K, d_Kyy, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kyz, n_K, d_Kyz, n_K );
	 stat = hipblasSetMatrix (n_K,n_K, sizeof (float), Kzz, n_K, d_Kzz, n_K );
	 
	//allocate the internal M and H vectors
	bytes = n_K * sizeof(float);
	hipMalloc( &d_Mx, bytes );
	hipMalloc( &d_My, bytes );
	hipMalloc( &d_Mz, bytes );

	hipMalloc( &d_Hx, bytes );
	hipMalloc( &d_Hy, bytes );
	hipMalloc( &d_Hz, bytes );
	
	//initialize the cuBlas handle
	hipblasCreate(&handle);
	 
 }
 
 /**
 Get the resulting demag field through
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kxy * Mx + Kyy * My + Kyz * Mz
 Hz = Kxz * Mx + Kyz * My + Kzz * Mz
 
 Noting that K is symmetric, i.e. Kxy = Kyx etc.
 
 @param Mx, My and Mz are n,1 arrays assuming that n == n_K (no checking, so use it correctly!)
 @param Hx, Hy and Hz are the output field vectors (also size n,1)
 @param pref is the factor that goes in front of the multiplication, i.e.
 Hx = pref * ( Kxx * Mx + Kxy * My + Kxz * Mz ) etc.
 */
 void cu_MVMult_GetH( const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, int* n, float* pref )
 {
	 hipblasStatus_t stat;
	 //copy the M vectors to the GPU card
	 stat = hipblasSetVector (n_K, sizeof (float), Mx, 1, d_Mx, 1);
	 stat = hipblasSetVector (n_K, sizeof (float), My, 1, d_My, 1);
	 stat = hipblasSetVector (n_K, sizeof (float), Mz, 1, d_Mz, 1);
	 
	 float beta = 0.;
	 
	 //Kxx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxx, n_K, d_Mx, 1, &beta, d_Hx, 1);
	 //Kxy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_My, 1, &beta, d_Hx, 1);
	 //Kxz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mz, 1, &beta, d_Hx, 1);
	 
	 beta = 0.;
	 //Kyx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_Mx, 1, &beta, d_Hy, 1);
	 //Kyy * My
	 beta = 1.; //change beta = 1 so that Hy is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyy, n_K, d_My, 1, &beta, d_Hy, 1);
	 //Kyz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_Mz, 1, &beta, d_Hy, 1);
	 
	 beta = 0.;
	 //Kzx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mx, 1, &beta, d_Hz, 1);
	 //Kzy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_My, 1, &beta, d_Hz, 1);
	 //Kzz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kzz, n_K, d_Mz, 1, &beta, d_Hz, 1);
	 
	 //copy the resulting field vector back	 
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hx, 1, Hx, 1);
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hy, 1, Hy, 1);
	 stat = hipblasGetVector (n_K, sizeof (float), d_Hz, 1, Hz, 1);
	 
 }
 
 void cu_test( const float* K, int* n)
 {
	 // FILE *fp;

	//fp = fopen("test.txt", "w+");
	for ( int i=0; i < *n; i++ )
		printf("%f %d\n", K[i], 12 );
	
	//fclose(fp);
 }
 
 void cu_destroy()
 {
	 if ( d_Kxx != NULL )
	 {
		 hipFree( d_Kxx );
		 d_Kxx = NULL;
		 hipFree( d_Kxy );
		 d_Kxy = NULL;
		 hipFree( d_Kxz );
		 d_Kxz = NULL;
		 hipFree( d_Kyy );
		 d_Kyy = NULL;
		 hipFree( d_Kyz );
		 d_Kyz = NULL;
		 hipFree( d_Kzz );
		 d_Kzz = NULL;
	 }
	 
	 if ( d_Hx != NULL )
	 {	 
		 hipFree( d_Hx );
		 d_Hx = NULL;
		 hipFree( d_Hy );
		 d_Hy = NULL;
		 hipFree( d_Hz );
		 d_Hz = NULL;
	 }
	 
	 if ( d_Mx != NULL )
	 {
		 hipFree( d_Mx );
		 d_Mx = NULL;
		 hipFree( d_My );
		 d_My = NULL;
		 hipFree( d_Mz );
		 d_Mz = NULL;
	 }
	 
	 if ( handle != NULL )
	{
		hipblasDestroy(handle);
		handle = NULL;
	}
	
		
	freeSparseMatrix( &spKxx );
	freeSparseMatrix( &spKxy );
	freeSparseMatrix( &spKxz );
	freeSparseMatrix( &spKyy );
	freeSparseMatrix( &spKyz );
	freeSparseMatrix( &spKzz );

	if ( sparse_descr != NULL )
	{
		hipsparseDestroyMatDescr(sparse_descr);
		sparse_descr = NULL;
	}

	if (sparse_handle != NULL )
	{
		hipsparseDestroy(sparse_handle);
		sparse_handle = NULL;
	}

 }



void freeSparseMatrix( CUSparse* mat )
{
	if ( mat->values != NULL )
	{
		hipFree( mat->values );
		mat->values = NULL;
		hipFree( mat->cols );
		mat->cols = NULL;
		hipFree( mat->rows );
		mat->rows = NULL;

	}
}
