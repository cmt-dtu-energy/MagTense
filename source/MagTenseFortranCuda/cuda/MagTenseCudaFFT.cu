#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <hipfft/hipfft.h>


// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

// These are the inline versions for all of the SDK helper functions
void __checkCudaErrors( int err, const char *file, const int line )
{
    if( 0 != err) {
		FILE *fp;

		fp = fopen("error.txt", "w+");
		fprintf(fp,"Error in file <%s>, line %i:  Error code %d\n", file, line, err );
		fclose(fp);
        exit(-1);
    }
}

 /**
 Demag matrices to be stored in GPU memory
 */
 float2* d_Kxx = NULL;
 float2* d_Kxy = NULL;
 float2* d_Kxz = NULL;
 float2* d_Kyy = NULL;
 float2* d_Kyz = NULL;
 float2* d_Kzz = NULL;
 
 //Magnetization vectors on GPU
 float2* d_Mx = NULL;
 float2* d_My = NULL;
 float2* d_Mz = NULL;
 
 //Magnetic field vectors on GPU
 float2* d_Hx = NULL;
 float2* d_Hy = NULL;
 float2* d_Hz = NULL;
 
 //Vector handles
 hipsparseDnVecDescr_t d_Mx_descr = NULL;
 hipsparseDnVecDescr_t d_My_descr = NULL;
 hipsparseDnVecDescr_t d_Mz_descr = NULL;
 
 hipsparseDnVecDescr_t d_Hx_descr = NULL;
 hipsparseDnVecDescr_t d_Hy_descr = NULL;
 hipsparseDnVecDescr_t d_Hz_descr = NULL;
 
 
 //size of the matrices (n_K x n_K)
 int n_K;
 
 /**
 Sparse demag matrices to be stored in the GPU memory
 */
 
 struct CUSparse
 {
	// api handle
	hipsparseSpMatDescr_t descr = NULL;
	//data values
	float* values = NULL;
	//column indices (size nnz)
	int* cols;
	//row indices (size n + 1  with the last element equal to nnz)
	int* rows;
	//no of matrix elements (matrices are n x n )
	int n;
 };
 
 CUSparse spKxx;
 CUSparse spKxy;
 CUSparse spKxz;
 CUSparse spKyy;
 CUSparse spKyz;
 CUSparse spKzz;
 
 //general handle for cuBlas (initalized once)
 hipblasHandle_t handle = NULL;
 
 
 //handle to the sparse matrix description
 hipsparseMatDescr_t sparse_descr = NULL;

 //handle to the sparse matrix blas
 hipsparseHandle_t sparse_handle = NULL; 
 
 void loadSparseToDevice( float* values, int* colInds, int* rowInds, CUSparse* mat, int nnz_ );
 void freeSparseMatrix( CUSparse* mat );
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, 2019
 Initializes the sparse matrices for later use in the matrix-vector multiplications
 n is the no. of rows and cols in the matrices
 nnz is the no. of non-zero elements
 mat_no 1...6 identifies which matrix to load (Kxx = 1, Kxy = 2, Kxz = 3, Kyy = 4, Kyz = 5, Kzz = 6 )
 values float array with the non-zero values of the matrix (length nnz)
 colInds int array with the column indices of the CSR stored matrix (length nnz)
 rowInds the indices of the first non-zero element in each row (size nrows+1 and the last element has to be equal to nnz + rowInds[0] )
 */
 void cu_initDemagMatrices_sparse( const int* n, const int* nnz, const int* mat_no, float* values, int* colInds, int* rowInds )
 {
	
	
	 //should only be done once
	 if ( sparse_handle == NULL )
	 {
		 //handle to the sparse matrix multiplier in CUDA
		checkCudaErrors(hipsparseCreate(&sparse_handle));
		
		//no. of rows and columns
		spKxx.n = *n;
		spKxy.n = *n;
		spKxz.n = *n;
		spKyy.n = *n;
		spKyz.n = *n;
		spKzz.n = *n;
		
		//allocate memory for the magnetization vectors
		checkCudaErrors(hipMalloc((void**) &d_Mx, *n * sizeof(float)));
		checkCudaErrors(hipMalloc((void**) &d_My, *n * sizeof(float)));
		checkCudaErrors(hipMalloc((void**) &d_Mz, *n * sizeof(float)));
		
		//allocate memory for the magnetic field vectors
		checkCudaErrors(hipMalloc((void**) &d_Hx, *n * sizeof(float)));
		checkCudaErrors(hipMalloc((void**) &d_Hy, *n * sizeof(float)));
		checkCudaErrors(hipMalloc((void**) &d_Hz, *n * sizeof(float)));
			 
		checkCudaErrors(hipsparseCreateMatDescr(&sparse_descr));
		checkCudaErrors(hipsparseSetMatType(sparse_descr,HIPSPARSE_MATRIX_TYPE_GENERAL));
		checkCudaErrors(hipsparseSetMatIndexBase(sparse_descr,HIPSPARSE_INDEX_BASE_ONE));
		
		
		//The handle on the device for the magnetization vectors
		checkCudaErrors(hipsparseCreateDnVec( &d_Mx_descr, *n, d_Mx, HIP_R_32F));
		checkCudaErrors(hipsparseCreateDnVec( &d_My_descr, *n, d_My, HIP_R_32F));
		checkCudaErrors(hipsparseCreateDnVec( &d_Mz_descr, *n, d_Mz, HIP_R_32F));
		
		//handles for the field on the device
		checkCudaErrors(hipsparseCreateDnVec( &d_Hx_descr, *n, d_Hx, HIP_R_32F));
		checkCudaErrors(hipsparseCreateDnVec( &d_Hy_descr, *n, d_Hy, HIP_R_32F));
		checkCudaErrors(hipsparseCreateDnVec( &d_Hz_descr, *n, d_Hz, HIP_R_32F));
		
		
	 }
	
	switch( *mat_no )
	{
		case 1:			
			loadSparseToDevice( values, colInds, rowInds, &spKxx, *nnz );

			break;
		case 2:
			loadSparseToDevice( values, colInds, rowInds, &spKxy, *nnz );

			break;
		case 3:
			loadSparseToDevice( values, colInds, rowInds, &spKxz, *nnz );

			break;
		case 4:
			loadSparseToDevice( values, colInds, rowInds, &spKyy, *nnz );

			break;
		case 5:
			loadSparseToDevice( values, colInds, rowInds, &spKyz, *nnz );

			break;
		case 6:
			loadSparseToDevice( values, colInds, rowInds, &spKzz, *nnz );
			
			break;
			
	}
	
	
	
 }
 
 
 void loadSparseToDevice( float* values, int* colInds, int* rowInds, CUSparse* mat, int nnz_ )
 {
	//allocate the row inds (+1 in size as the last element contains nnz + rowInds(0)
	checkCudaErrors(hipMalloc((void**) &(mat->rows), (mat->n + 1) * sizeof(int)));
	//allocate the column inds
	checkCudaErrors(hipMalloc((void**) &(mat->cols), nnz_ * sizeof(int)));
	//allocate the values array
	checkCudaErrors(hipMalloc((void**) &(mat->values), nnz_ * sizeof(float)));
	
	//copy to device
	checkCudaErrors(hipMemcpy( mat->rows, rowInds, (mat->n + 1) * sizeof(int),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy( mat->cols, colInds, nnz_ * sizeof(int),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy( mat->values, values, nnz_ * sizeof(float),hipMemcpyHostToDevice));
	
	//init the sparse matrix handles
	checkCudaErrors(hipsparseCreateCsr(&(mat->descr), mat->n, mat->n, nnz_, mat->rows, mat->cols, mat->values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
	
 }

 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, 2019
 Does the matrix-vector product of a sparse matrix and a dense vector
 and returns the demag field on the form:
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kyz * Mx + Kyy * My + Kzz * Mz
 Hz = Kzx * Mx + Kzy * My + Kzz * Mz
 
 */
 void cu_MVMult_GetH_sparse(const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, const float* pref)
 {	 
	int n = spKxx.n;
	
	//Possible extra memory needed for CUDA operations
	size_t bufferSize = 0;
	void *dBuffer = NULL;
	
	//Copy Mx, My and Mz to device memory	
	checkCudaErrors(hipMemcpy( d_Mx, Mx, n * sizeof(float), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_My, My, n * sizeof(float), hipMemcpyHostToDevice ));
	checkCudaErrors(hipMemcpy( d_Mz, Mz, n * sizeof(float), hipMemcpyHostToDevice ));
	
	float alpha = *pref;
	//Hx = Kxx * Mx + Kxy * My + Kxz * Mz	
	float beta = 0.0;
	//by setting beta = 0 in the first call we ensure that the previous values of d_Hx are irrelevant
	//as this operation does the following:
	//d_Hx = alpha * d_Kxx * d_Mx + beta * d_Hx
	//Kxx * Mx
	checkCudaErrors(hipsparseSpMV_bufferSize( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
								 &alpha, spKxx.descr, d_Mx_descr, &beta, d_Hx_descr, HIP_R_32F, 
								 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
	checkCudaErrors(hipMalloc( &dBuffer, bufferSize ));
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKxx.descr, d_Mx_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
			
	beta = 1.0;
	//Kxy * My
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKxy.descr, d_My_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
								 
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKxz.descr, d_Mz_descr, &beta, d_Hx_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));

			   
	//Hy = Kxy * Mx + Kyy * My + Kyz * Mz	
	beta = 0.;
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKxy.descr, d_Mx_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
			
	beta = 1.0;	
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKyy.descr, d_My_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
								 
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKyz.descr, d_Mz_descr, &beta, d_Hy_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
								 
	//Hz = Kzx * Mx + Kzy * My + Kzz * Mz		
	beta = 0.;
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKxz.descr, d_Mx_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
			
	beta = 1.0;
	//Kxy * My
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKyz.descr, d_My_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
								 
	checkCudaErrors(hipsparseSpMV( sparse_handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, spKzz.descr, d_Mz_descr, &beta, d_Hz_descr, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
			   
	//copy the solution back
	checkCudaErrors(hipMemcpy( Hx, d_Hx, n * sizeof(float), hipMemcpyDeviceToHost ));
	checkCudaErrors(hipMemcpy( Hy, d_Hy, n * sizeof(float), hipMemcpyDeviceToHost ));
	checkCudaErrors(hipMemcpy( Hz, d_Hz, n * sizeof(float), hipMemcpyDeviceToHost ));
	
	if ( dBuffer != NULL )
	{
		hipFree( dBuffer );
		dBuffer = NULL;
	}
 }
 
 
 /**
 Kaspar K. Nielsen, kasparkn@gmail.com.dk, DTU 2019
 Method for transfering the demagnetization matrices from Fortran
 @param Kxx, Kxy, Kxz, Kyy, Kyz and Kzz are 2D arrays of size n x n implemented as an array of n pointers each pointing to an array of n values
 @param n is the size of the square matrix
 */
 void cu_initDemagMatrices( const float* Kxx, const float* Kxy, const float* Kxz, const float* Kyy, const float* Kyz, const float* Kzz, int* n)
 {	 
	/* FILE *fp;

	fp = fopen("test.txt", "w+");
	fprintf(fp,"%f %d\n", Kxx[0], *n );
	//fprintf(fp, "This is testing for fprintf...\n");
	//fputs("This is testing for fputs...\n", fp);
	fclose(fp);
	 */
	 
	 n_K = *n;
	 
	 //Allocate the device (GPU) arrays
	 size_t bytes = n_K * n_K * sizeof(float);
	 
	 hipMalloc( &d_Kxx, bytes );
	 hipMalloc( &d_Kxy, bytes );
	 hipMalloc( &d_Kxz, bytes );
	 hipMalloc( &d_Kyy, bytes );
	 hipMalloc( &d_Kyz, bytes );
	 hipMalloc( &d_Kzz, bytes );
	 
	 
	 
	 //copy the demag tensors to the device	 
	 checkCudaErrors(hipblasSetMatrix (n_K,n_K, sizeof (float), Kxx, n_K, d_Kxx, n_K ));
	 checkCudaErrors(hipblasSetMatrix (n_K,n_K, sizeof (float), Kxy, n_K, d_Kxy, n_K ));
	 checkCudaErrors(hipblasSetMatrix (n_K,n_K, sizeof (float), Kxz, n_K, d_Kxz, n_K ));
	 checkCudaErrors(hipblasSetMatrix (n_K,n_K, sizeof (float), Kyy, n_K, d_Kyy, n_K ));
	 checkCudaErrors(hipblasSetMatrix (n_K,n_K, sizeof (float), Kyz, n_K, d_Kyz, n_K ));
	 checkCudaErrors(hipblasSetMatrix (n_K,n_K, sizeof (float), Kzz, n_K, d_Kzz, n_K ));
	 
	//allocate the internal M and H vectors
	bytes = n_K * sizeof(float);
	hipMalloc( &d_Mx, bytes );
	hipMalloc( &d_My, bytes );
	hipMalloc( &d_Mz, bytes );

	hipMalloc( &d_Hx, bytes );
	hipMalloc( &d_Hy, bytes );
	hipMalloc( &d_Hz, bytes );
	
	//initialize the cuBlas handle
	hipblasCreate(&handle);
	 
 }
 
 /**
 Get the resulting demag field through
 Hx = Kxx * Mx + Kxy * My + Kxz * Mz
 Hy = Kxy * Mx + Kyy * My + Kyz * Mz
 Hz = Kxz * Mx + Kyz * My + Kzz * Mz
 
 Noting that K is symmetric, i.e. Kxy = Kyx etc.
 
 @param Mx, My and Mz are n,1 arrays assuming that n == n_K (no checking, so use it correctly!)
 @param Hx, Hy and Hz are the output field vectors (also size n,1)
 @param pref is the factor that goes in front of the multiplication, i.e.
 Hx = pref * ( Kxx * Mx + Kxy * My + Kxz * Mz ) etc.
 */
 void cu_MVMult_GetH( const float* Mx, const float* My, const float* Mz, float* Hx, float* Hy, float* Hz, int* n, float* pref )
 {
	 //copy the M vectors to the GPU card
	 checkCudaErrors(hipblasSetVector (n_K, sizeof (float), Mx, 1, d_Mx, 1));
	 checkCudaErrors(hipblasSetVector (n_K, sizeof (float), My, 1, d_My, 1));
	 checkCudaErrors(hipblasSetVector (n_K, sizeof (float), Mz, 1, d_Mz, 1));
	 
	 float beta = 0.;
	 
	 //Kxx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxx, n_K, d_Mx, 1, &beta, d_Hx, 1);
	 //Kxy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_My, 1, &beta, d_Hx, 1);
	 //Kxz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mz, 1, &beta, d_Hx, 1);
	 
	 beta = 0.;
	 //Kyx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxy, n_K, d_Mx, 1, &beta, d_Hy, 1);
	 //Kyy * My
	 beta = 1.; //change beta = 1 so that Hy is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyy, n_K, d_My, 1, &beta, d_Hy, 1);
	 //Kyz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_Mz, 1, &beta, d_Hy, 1);
	 
	 beta = 0.;
	 //Kzx * Mx
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kxz, n_K, d_Mx, 1, &beta, d_Hz, 1);
	 //Kzy * My
	 beta = 1.; //change beta = 1 so that Hx is updated and not overwritten
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kyz, n_K, d_My, 1, &beta, d_Hz, 1);
	 //Kzz * Mz
	 hipblasSgemv(handle, HIPBLAS_OP_N, n_K, n_K, pref, d_Kzz, n_K, d_Mz, 1, &beta, d_Hz, 1);
	 
	 //copy the resulting field vector back	 
	 checkCudaErrors(hipblasGetVector (n_K, sizeof (float), d_Hx, 1, Hx, 1));
	 checkCudaErrors(hipblasGetVector (n_K, sizeof (float), d_Hy, 1, Hy, 1));
	 checkCudaErrors(hipblasGetVector (n_K, sizeof (float), d_Hz, 1, Hz, 1));
	 
 }
 
 void cu_test( const float* K, int* n)
 {
	 // FILE *fp;

	//fp = fopen("test.txt", "w+");
	for ( int i=0; i < *n; i++ )
		printf("%f %d\n", K[i], 12 );
	
	//fclose(fp);
 }
 
 void cu_destroy()
 {
	if ( d_Kxx != NULL )
	{
		checkCudaErrors(hipFree( d_Kxx ));
		d_Kxx = NULL;
		checkCudaErrors(hipFree( d_Kxy ));
		d_Kxy = NULL;
		checkCudaErrors(hipFree( d_Kxz ));
		d_Kxz = NULL;
		checkCudaErrors(hipFree( d_Kyy ));
		d_Kyy = NULL;
		checkCudaErrors(hipFree( d_Kyz ));
		d_Kyz = NULL;
		checkCudaErrors(hipFree( d_Kzz ));
		d_Kzz = NULL;
	}
	 
	if ( d_Hx != NULL )
	{	 
		checkCudaErrors(hipFree( d_Hx ));
		d_Hx = NULL;
		checkCudaErrors(hipFree( d_Hy ));
		d_Hy = NULL;
		checkCudaErrors(hipFree( d_Hz ));
		d_Hz = NULL;
	}
	 
	if ( d_Mx != NULL )
	{
		checkCudaErrors(hipFree( d_Mx ));
		d_Mx = NULL;
		checkCudaErrors(hipFree( d_My ));
		d_My = NULL;
		checkCudaErrors(hipFree( d_Mz ));
		d_Mz = NULL;
	}
	 
	if ( d_Hx_descr )
	{	 
		checkCudaErrors(hipsparseDestroyDnVec( d_Hx_descr ));
		checkCudaErrors(hipsparseDestroyDnVec( d_Hy_descr ));
		checkCudaErrors(hipsparseDestroyDnVec( d_Hz_descr ));
	}
	 
	if ( d_Mx_descr )
	{
		checkCudaErrors(hipsparseDestroyDnVec( d_Mx_descr ));
		checkCudaErrors(hipsparseDestroyDnVec( d_My_descr ));
		checkCudaErrors(hipsparseDestroyDnVec( d_Mz_descr ));
	}
	 
	if ( handle != NULL )
	{
		checkCudaErrors(hipblasDestroy(handle));
		handle = NULL;
	}

		
	freeSparseMatrix( &spKxx );
	freeSparseMatrix( &spKxy );
	freeSparseMatrix( &spKxz );
	freeSparseMatrix( &spKyy );
	freeSparseMatrix( &spKyz );
	freeSparseMatrix( &spKzz );	

	if ( sparse_descr != NULL )
	{
		checkCudaErrors(hipsparseDestroyMatDescr(sparse_descr));
		sparse_descr = NULL;
	}

	if (sparse_handle != NULL )
	{
		checkCudaErrors(hipsparseDestroy(sparse_handle));
		sparse_handle = NULL;
	}
 }

void freeSparseMatrix( CUSparse* mat )
{
	if ( mat->values != NULL )
	{
		checkCudaErrors(hipsparseDestroySpMat( mat->descr ));
		checkCudaErrors(hipFree( mat->values ));
		mat->values = NULL;
		checkCudaErrors(hipFree( mat->cols ));
		mat->cols = NULL;
		checkCudaErrors(hipFree( mat->rows ));
		mat->rows = NULL;
	}
}