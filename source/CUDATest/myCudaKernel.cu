
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}
 
float* internalBuffer;
int nBuf;

void cu_initBuf( float* buf, int* n)
{
	nBuf = *n;
	
	internalBuffer = (float*)malloc(nBuf*sizeof(float));
	
	for ( int i = 0; i < nBuf; i++ )
	{
		internalBuffer[i] = buf[i];
	}
}
 
 void cu_destroy()
 {
	free(internalBuffer);
 }
 
void cu_vecAdd_wrapper(float* val, int* nn)
{
	for ( int uu=0; uu < *nn; uu++ )
	{
		val[uu] = internalBuffer[uu] * 3;
	}
		
    // Size of vectors
    int n = 100000;
 
    // Host input vectors
    float *h_a;
    float *h_b;
    //Host output vector
    float *h_c;
 
    // Device input vectors
    float *d_a;
    float *d_b;
    //Device output vector
    float *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);
 
    // Allocate memory for each vector on host
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
 
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sinf(i)*sinf(i);
        h_b[i] = cosf(i)*cosf(i);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Sum up vector c and print result divided by n, this should equal 1 within error
    float sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/n);
 	
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
	

}